#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <math.h>
#include <algorithm>
#include ""

#define ROWLEN 5
#define COLLEN 5
#define MAX 10

//N = arraySize
//#define n 512
//#define  multiple 3
#define N 1024

//S = threadNum
#define S 32


using namespace std;

//Use the GPU to calculate the KNN's answer
__global__ void getDistanceGPU(double trainSet[ROWLEN][COLLEN], double* testData, double* dis)
{

    int xid = threadIdx.x + blockIdx.x * blockDim.x;
    int yid = threadIdx.y + blockIdx.y * blockDim.y;

    int row = yid;
    int col = xid;

    printf("%f!!! ", dis[col]);

    if (col < ROWLEN && row < COLLEN)
    {
        for (int i = 0; i < COLLEN; i++)
        {
            double temp = 0;
            temp = pow((trainSet[row][i] - testData[i]), 2);
            dis[col] += sqrt(temp);
        }
    }
    
}

void gpuCal(double* a[ROWLEN], double b[ROWLEN], double c[COLLEN])
{
    double (*dev_a)[ROWLEN];
    double *dev_b;
    double *dev_c;

    //在GPU中开辟空间
    hipMalloc((void**)&dev_a, ROWLEN * COLLEN * sizeof(double));
    hipMalloc((void**)&dev_b, ROWLEN * sizeof(double));
    hipMalloc((void**)&dev_c, COLLEN * sizeof(double));

    //将CPU内容复制到GPU
    hipMemcpy(dev_a, a, ROWLEN * COLLEN * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, ROWLEN * sizeof(double), hipMemcpyHostToDevice);

    //声明时间Event
    float time = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //GPU开始计算
    dim3 threadsPerBlock(S, S);
    dim3 blocksPerGrid(ROWLEN / threadsPerBlock.x, COLLEN / threadsPerBlock.y);
    getDistanceGPU << <blocksPerGrid, threadsPerBlock >> > (dev_a, dev_b, dev_c);
    //结束计时
    hipEventRecord(stop, 0);

    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    //计算时间差
    hipEventElapsedTime(&time, start, stop);
    //将内容拷贝回CPU
    hipMemcpy(c, dev_c, COLLEN * sizeof(double), hipMemcpyDeviceToHost);

    for (int j = 0; j < COLLEN; j++)
    {
            printf("%f ", c[j]);
    }
    printf("\n");
    printf("GPU: spendTime: %fms\n\n\n", time);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}

//Calculate the distance between testData and dataSet[i]
double getDistance(double* d1, int* d2);

//calculate all the distance between testData and each training data
void getAllDistance(double** trainSet, double* testData, double* discard_block);

// Randomly generated training set 
void randNum(double** trainSet, int rlen, int clen);

//Randomly generated testDate
void randNum(double* testData, int clen);

//Print the trainSet
void print(double** trainSet, int rlen, int clen);

//Print the testSet
void print(double* testData, int clen);

int main(int argc, char const* argv[])
{
    double** trainSet;
    double* testData;
    double* dis;
    trainSet = new double* [ROWLEN];
    testData = new double[COLLEN];
    dis = new double[ROWLEN];
    randNum(trainSet, ROWLEN, COLLEN);
    randNum(testData, COLLEN);

    gpuCal(trainSet, testData, dis);

    cout << "-----------------trainSet----------------------------" << endl;
    print(trainSet, ROWLEN, COLLEN);
    cout << "-----------------testSet----------------------------" << endl;
    print(testData, COLLEN);
    cout << "-----------------dis-------------------------------" << endl;
    print(dis, COLLEN);
    sort(dis, dis + COLLEN);
    print(dis, COLLEN);
    return 0;
}


//Calculate the distance between trainSet and testData
double getDistance(double* d1, double* d2)
{
    double dis = 0;
    for (int i = 0; i < COLLEN; i++)
    {
        dis += pow((d1[i] - d2[i]), 2);
    }
    return sqrt(dis);
}

//calculate all the distance between testData and each training data
void getAllDistance(double** trainSet, double* testData, double* dis)
{
    for (int i = 0; i < ROWLEN; i++)
    {
        dis[i] = getDistance(trainSet[i], testData);
    }
}

// Randomly generated training set 
void randNum(double** trainSet, int rlen, int clen)
{
    for (int i = 0; i < rlen; i++)
    {
        trainSet[i] = new double[clen];
    }
    for (int i = 0; i < rlen; i++)
    {
        for (int j = 0; j < clen; j++)
        {
            trainSet[i][j] = rand() % MAX;
        }
    }
}

//Randomly generated testDatd
void randNum(double* testData, int clen)
{
    for (int i = 0; i < clen; i++)
    {
        testData[i] = rand() % MAX;
    }
}
//Print the trainSet
void print(double** trainSet, int rlen, int clen)
{
    for (int i = 0; i < rlen; i++)
    {
        for (int j = 0; j < clen; j++)
        {
            cout << trainSet[i][j] << " ";
        }
        cout << endl;
    }
}

//Print the testSet
void print(double* testData, int clen)
{
    for (int i = 0; i < clen; i++)
    {
        cout << testData[i] << " ";
    }
    cout << endl;
}


